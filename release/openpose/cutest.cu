
#include <hip/hip_runtime.h>



#define N 5

__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N)
		C[i][j] = A[i][j] + B[i][j];
}

