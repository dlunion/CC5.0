#include "hip/hip_runtime.h"
#include "caffe/layers/triplet_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
void TripletLossLayer<Dtype>::ComputeDiff_gpu(const Dtype *x_1,
  const Dtype *x_2, const Dtype x_1_norm, const Dtype x_2_norm,
  const Dtype inner_val, Dtype *x_1_diff) {
  caffe_gpu_scale(feature_dim_, Dtype(1) / (x_1_norm * x_2_norm),
      x_2, x_1_diff);
  Dtype x_1_norm_cubic = x_1_norm * x_1_norm * x_1_norm;
  caffe_gpu_axpby(feature_dim_, -inner_val / (x_1_norm_cubic * x_2_norm),
      x_1, Dtype(1), x_1_diff);
}

template <typename Dtype>
void TripletLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
  Dtype eps = this->layer_param_.triplet_loss_param().eps();
  Dtype loss = 0;
  Dtype margin = this->layer_param_.triplet_loss_param().margin();
  caffe_gpu_gemm(CblasNoTrans, CblasTrans, sample_num_, sample_num_,
      feature_dim_, Dtype(1), bottom[0]->gpu_data(),
      bottom[0]->gpu_data(), Dtype(0),
      inner_matrix_.mutable_gpu_data());

  for (int i = 0; i < triplet_num_; ++i) {
    int a_idx = bottom[1]->cpu_data()[i * 3];
    int p_idx = bottom[1]->cpu_data()[i * 3 + 1];
    int n_idx = bottom[1]->cpu_data()[i * 3 + 2];
    const Dtype *a_pointer = bottom[0]->gpu_data() + a_idx * feature_dim_;
    const Dtype *p_pointer = bottom[0]->gpu_data() + p_idx * feature_dim_;
    const Dtype *n_pointer = bottom[0]->gpu_data() + n_idx * feature_dim_;
    const Dtype *inner_matrix_data = inner_matrix_.cpu_data();
    Dtype a_norm = sqrt(inner_matrix_data[a_idx * sample_num_ + a_idx] + eps);
    Dtype p_norm = sqrt(inner_matrix_data[p_idx * sample_num_ + p_idx] + eps);
    Dtype n_norm = sqrt(inner_matrix_data[n_idx * sample_num_ + n_idx] + eps);
    Dtype inner_ap = inner_matrix_data[a_idx * sample_num_ + p_idx];
    Dtype inner_an = inner_matrix_data[a_idx * sample_num_ + n_idx];
    Dtype dist_ap = inner_ap / (a_norm * p_norm);
    Dtype dist_an = inner_an / (a_norm * n_norm);
    if (dist_ap - dist_an - margin < 0) {
      ComputeDiff_gpu(a_pointer, p_pointer, a_norm,
          p_norm, inner_ap, diff_ap_.mutable_gpu_data());
      ComputeDiff_gpu(a_pointer, n_pointer, a_norm,
          n_norm, inner_an, diff_an_.mutable_gpu_data());
      ComputeDiff_gpu(p_pointer, a_pointer, p_norm,
          a_norm, inner_ap, diff_pa_.mutable_gpu_data());
      ComputeDiff_gpu(n_pointer, a_pointer, n_norm,
          a_norm, inner_an, diff_na_.mutable_gpu_data());

      caffe_gpu_axpby(feature_dim_, Dtype(1),
          diff_an_.gpu_data(), Dtype(1),
          bottom_diff_.mutable_gpu_data() + (a_idx * feature_dim_));
      caffe_gpu_axpby(feature_dim_, Dtype(-1),
          diff_ap_.gpu_data(), Dtype(1),
          bottom_diff_.mutable_gpu_data() + (a_idx * feature_dim_));
      caffe_gpu_axpby(feature_dim_, Dtype(-1),
          diff_pa_.gpu_data(), Dtype(1),
          bottom_diff_.mutable_gpu_data() + (p_idx * feature_dim_));
      caffe_gpu_axpby(feature_dim_, Dtype(1),
          diff_na_.gpu_data(), Dtype(1),
          bottom_diff_.mutable_gpu_data() + (n_idx * feature_dim_));

      loss += dist_an + margin - dist_ap;
    }
  }
  //Dtype scalar = Dtype(1) / triplet_num_;
  Dtype scalar = Dtype(1) / sample_num_;
  top[0]->mutable_cpu_data()[0] = loss * scalar;
}

template <typename Dtype>
void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    //Dtype scalar = top[0]->cpu_diff()[0] / triplet_num_;
    Dtype scalar = top[0]->cpu_diff()[0] / sample_num_;
    caffe_gpu_scale(bottom_diff_.count(), scalar, bottom_diff_.gpu_data(),
        bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}
