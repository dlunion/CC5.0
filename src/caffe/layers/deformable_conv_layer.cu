#include <vector>
#include <iostream>

#include "caffe/layers/deformable_conv_layer.hpp"

using namespace std;
namespace caffe {
template <typename Dtype>
void DeformableConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* weights = this->blobs_[0]->gpu_data();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* offset = bottom[1]->gpu_data();
    top[0]->scale_data(0);//data protect
    Dtype* top_data = top[0]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      const Dtype* col_buff = bottom_data + n*this->bottom_dim_;
      deformable_im2col_gpu<Dtype>(bottom_data + n*this->bottom_dim_, //data_col
                                          offset + n*this->input_offset_dim_,//offset
                                          bottom[0]->shape(1),
                                          bottom[0]->shape(2),bottom[0]->shape(3),this->kernel_shape_.cpu_data()[0],this->kernel_shape_.cpu_data()[1],
                                          this->pad_.cpu_data()[0],this->pad_.cpu_data()[1],this->stride_.cpu_data()[0],this->stride_.cpu_data()[1],
                                          this->dilation_.cpu_data()[0],this->dilation_.cpu_data()[1],this->deformable_group_,
                                          this->col_buffer_.mutable_gpu_data());

    // gemm
    for (int g = 0; g < this->group_; ++g) {
          caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, this->conv_out_channels_ /
                                this->group_, this->conv_out_spatial_dim_, this->kernel_dim_,
                                (Dtype)1., weights + this->weight_offset_ * g, this->col_buffer_.gpu_data() + this->col_offset_ * g,
                                (Dtype)0., top[0]->mutable_gpu_data() + n * this->top_dim_ + this->output_offset_ * g);                      
       
    }
    
    if (this->bias_term_) {
      const Dtype* bias = this->blobs_[1]->gpu_data();
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, this->num_output_,
          this->out_spatial_dim_, 1, (Dtype)1., bias, this->bias_multiplier_.gpu_data(),
          (Dtype)1., top_data + n * this->top_dim_);
      }
  }
  
  
}

template <typename Dtype>
void DeformableConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
      const Dtype* weight = this->blobs_[0]->gpu_data();
      Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
      const Dtype* top_diff = top[0]->gpu_diff();
     // Bias gradient, if necessary.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
        for (int n = 0; n < this->num_; ++n) {
          caffe_gpu_gemv<Dtype>(CblasNoTrans, this->num_output_, this->out_spatial_dim_, 1.,
            top_diff + n * this->top_dim_ , this->bias_multiplier_.gpu_data(), 1., bias_diff);
        }
      }
      if (this->param_propagate_down_[0] || propagate_down[0]) {
        const Dtype* bottom_data = bottom[0]->gpu_data();
        bottom[0]->scale_diff(0);//data protect
        Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
        const Dtype * bottom_offset=bottom[1]->gpu_data();
        bottom[1]->scale_diff(0);//data protect
        Dtype * bottom_offset_diff = bottom[1]->mutable_gpu_diff();

        for (int n = 0; n < this->num_; ++n) {
          if (this->param_propagate_down_[0]) {            
              deformable_im2col_gpu<Dtype>(
                   bottom_data + n*this->bottom_dim_, //data_col
                    bottom_offset + n*this->input_offset_dim_,//offset
                    bottom[0]->shape(1),bottom[0]->shape(2),bottom[0]->shape(3),
                    this->kernel_shape_.cpu_data()[0],this->kernel_shape_.cpu_data()[1],
                    this->pad_.cpu_data()[0],this->pad_.cpu_data()[1],this->stride_.cpu_data()[0],this->stride_.cpu_data()[1],
                    this->dilation_.cpu_data()[0],this->dilation_.cpu_data()[1],this->deformable_group_,
                    this->col_buffer_.mutable_gpu_data());  
              for (int g = 0; g < this->group_; ++g) {
                      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, this->conv_out_channels_ / this->group_,
                              this->kernel_dim_, this->conv_out_spatial_dim_,
                              (Dtype)1.,  top[0]->gpu_diff() + this->output_offset_ * g, this->col_buffer_.gpu_data()+ this->col_offset_ * g,
                              (Dtype)1., weight_diff + this->weight_offset_ * g);
                              }
          }


          if (propagate_down[0]) {
            for (int g = 0; g < this->group_; ++g) {
              caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, this->kernel_dim_,
               this->conv_out_spatial_dim_, this->conv_out_channels_ / this->group_,
               (Dtype)1., weight + this->weight_offset_ * g,  top[0]->gpu_diff() + this->output_offset_ * g,
               (Dtype)0., this->col_buffer_.mutable_gpu_data() + this->col_offset_ * g);}
            // gradient w.r.t. input offset
              deformable_col2im_coord_gpu<Dtype>(this->col_buffer_.gpu_data(),
                bottom_data + n*this->bottom_dim_,
                bottom_offset + n*this->input_offset_dim_,
                this->col_buffer_.shape(0),bottom[0]->shape(2),bottom[0]->shape(3),
                this->kernel_shape_.cpu_data()[0],this->kernel_shape_.cpu_data()[1],
                  this->pad_.cpu_data()[0],this->pad_.cpu_data()[1],
                  this->stride_.cpu_data()[0],this->stride_.cpu_data()[1],
                  this->dilation_.cpu_data()[0],this->dilation_.cpu_data()[1],
                  this->deformable_group_, bottom_offset_diff + n*this->input_offset_dim_);
              // gradient w.r.t. input data
              deformable_col2im_gpu<Dtype>(this->col_buffer_.gpu_data(),
                bottom_offset + n*this->input_offset_dim_,
                this->conv_in_channels_,this->conv_input_shape_.cpu_data()[1],this->conv_input_shape_.cpu_data()[2],this->col_buffer_.shape(0),
                this->kernel_shape_.cpu_data()[0],this->kernel_shape_.cpu_data()[1],
                this->pad_.cpu_data()[0],this->pad_.cpu_data()[1],this->stride_.cpu_data()[0],this->stride_.cpu_data()[1],
                        this->dilation_.cpu_data()[0],this->dilation_.cpu_data()[1],this->deformable_group_,bottom_diff+ n*this->input_offset_dim_);
   
              }

      }    
  }
 }
//

INSTANTIATE_LAYER_GPU_FUNCS(DeformableConvolutionLayer);


}  // namespace caffe
